#include "hip/hip_runtime.h"
#ifndef _KNN_KERNEL_H_
#define _KNN_KERNEL_H_

#include "config.h"

////////////////////////////////////////////////////////////////////////////////
//! �������Ӿ��벢��С�ھ���ƽ��С��radius_square�������������������
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel( float* g_idata, float* g_odata) 
{
	const int step = gridDim.x * blockDim.x;					//����ѭ������
	int base_index = blockIdx.x * blockDim.x + threadIdx.x;		//�������ӵ����
	float value_sum;											//���������֮��
	float square;												//����֮������ƽ��
	float axis_diff;											//����֮������֮��

	unsigned int dim_i;
	
	float obj_data[particle_dimension];

	for(;base_index < particle_num; base_index += step)
	{
		value_sum = 0;

		//��globle memory���һ�����ӵ���Ϣ������obj_data��ȥ�����ڼ���õ����������ӵľ���
		for(int i=0; i<particle_dimension; i++)
			obj_data[i] = g_idata[base_index*data_dimension+i];
		
		//����share memoryʤ�ڴ�����е���Ϣ������ֻ�ܴ��thread_num���㣬��Ҫparticle_num/thread_num���̵߳���
		__shared__ float sdata[thread_num*data_dimension];

		for(int g=0;g<particle_num/thread_num;++g)
		{
			//��������Ϣ������share memory��ȥ
			for(int i=0; i<data_dimension; i++)
				sdata[threadIdx.x*data_dimension+i] = g_idata[data_dimension*thread_num*g+threadIdx.x*data_dimension+i];

			__syncthreads();

			for(int e=0;e<thread_num;++e)			
			{	
				square = 0;

				//������������������֮�����
				for(dim_i=0;dim_i<particle_dimension;++dim_i)
				{
					axis_diff = obj_data[dim_i]-sdata[e*4+dim_i];
					square += axis_diff * axis_diff;
				}

				//�ȽϾ���
				if(square < radius_square)
				{
					value_sum += sdata[e*(particle_dimension+1) + particle_dimension];//�������������
				}
			}
			__syncthreads();
		}

		g_odata[base_index]=value_sum;//����Ϣ�ŵ�Ŀ��globe memory���������ȥ
	}
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
