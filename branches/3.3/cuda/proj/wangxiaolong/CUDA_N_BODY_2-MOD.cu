#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAX 1000000
/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;
	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
/************************************************************************/
__global__ void MEMSET(int size,int* arr,int value, int threadnum, int blocknum) {
	int i,tid,bid,id,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=bid*threadnum+tid;
	step=ceil(float(size)/threadnum/blocknum);
	for(i=step*id;i<size && i<step*(id+1);i++) arr[i]=value;
	__syncthreads();
}
__device__ void device_MEMSET(int size,int* arr,int value,int tid, int threadnum) {
	int i,step;
	step=ceil(float(size)/threadnum);
	for(i=step*tid;i<size && i<step*(tid+1);i++) arr[i]=value;
	__syncthreads();
}
__global__ void SORT_1(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP, int threadnum, int blocknum, int sidenum, float r) {
	int i,tid,bid,id,IDx,IDy,IDz,ID,beg,end,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	step=ceil(float(N)/blocknum/threadnum);
	beg=step*id;
	end=beg+step;
	for(i=beg;i<N && i<end;i++) {
		IDx=oldP[i].x/r;	IDy=oldP[i].y/r;	IDz=oldP[i].z/r;
		ID=IDx+IDy*sidenum+IDz*sidenum*sidenum;
		belong[i]=ID;	begin[ID]++;	count[ID]++;
	}
}
__global__ void SORT_2(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP, int threadnum, int cubenum) {
	int i,tid,bid,id;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	if(id==0) for(i=1;i<cubenum;i++) begin[i]+=begin[i-1];
}
__global__ void SORT_3(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP, int threadnum, int blocknum) {
	int i,j,tid,bid,id,beg,end,step,ID;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	step=ceil(float(N)/blocknum/threadnum);
	beg=step*id;
	end=beg+step;
	for(i=beg;i<end;i++) { 
		ID=belong[i];
		j=--begin[ID];
		retrive[j]=i;
		newP[j]=oldP[i];
	}
}
void SORT(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP, int threadnum, int blocknum, int sidenum, int cubenum, float r) {
	MEMSET<<<blocknum,threadnum>>>(cubenum,begin,0, threadnum, blocknum);
	MEMSET<<<blocknum,threadnum>>>(cubenum,count,0, threadnum, blocknum);
	SORT_1<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP, threadnum, blocknum, sidenum, r);
	SORT_2<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP, threadnum, cubenum);
	SORT_3<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP, threadnum, blocknum);
}
//make sure that threadnum*blocknum equals to the number of total particals
__global__ void CALC(float4* newP,float* ans,int* count,int* begin,int gid, int threadnum, int blocknum, int sidenum, float r, float R) {
	int i,j,tid,bid,id,step;
	__shared__ float max_x,max_y,max_z,min_x,min_y,min_z;
	float sum=0;
	__shared__ int xid,Xid,yid,Yid,zid,Zid,readnum,totalnum;
	int access,_x,_y,_z;
	float4 _POINT;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=gid*blocknum*threadnum+bid*threadnum+tid;
	max_x=max_y=max_z=-1, min_x=min_y=min_z=1;
	__syncthreads();
	//every threads calculate one partical
	//figure out the perimeter of the zone in the block
	_POINT=newP[id];
	if(max_x<_POINT.x) max_x=_POINT.x;
	if(max_y<_POINT.y) max_y=_POINT.y;
	if(max_z<_POINT.z) max_z=_POINT.z;
	if(min_x>_POINT.x) min_x=_POINT.x;
	if(min_y>_POINT.y) min_y=_POINT.y;
	if(min_z>_POINT.z) min_z=_POINT.z;
	if(min_x-R>0) min_x-=R;	else min_x=0;
	if(min_y-R>0) min_y-=R;	else min_y=0;
	if(min_z-R>0) min_z-=R; else min_z=0;
	if(max_x+R<1) max_x+=R;	else max_x=0.99998;
	if(max_y+R<1) max_y+=R;	else max_y=0.99998;
	if(max_z+R<1) max_z+=R; else max_z=0.99998;
	xid=min_x/r,yid=min_y/r,zid=min_z/r;
	Xid=max_x/r,Yid=max_y/r,Zid=max_z/r;
	__syncthreads();
	readnum=(Xid-xid+1)*(Yid-yid+1)*(Zid-zid+1);
	__shared__ float4 M[MAX];
	__shared__ int BEG[MAX];
	device_MEMSET(readnum,BEG,0,tid, threadnum);
	if(tid==0) 	{
		for(i=0;i<readnum;i++) {
			_z=int(i/(Xid-xid+1)/(Yid-yid+1))+xid;
			_y=int((i-_z*(Xid-xid+1)*(Yid-yid+1))/(Xid-xid+1))+yid;
			_x=i%(Xid-xid+1)+xid;
			access=_x+_y*sidenum+_z*sidenum*sidenum;
			BEG[i+1]=BEG[i]+count[access];
		}
		totalnum=BEG[readnum];
	}
	__syncthreads();
	step=ceil(float(readnum)/threadnum);
	for(i=tid*step;i<readnum && i<(tid+1)*step;i++) {
		_z=int(i/(Xid-xid+1)/(Yid-yid+1))+xid;
		_y=int((i-_z*(Xid-xid+1)*(Yid-yid+1))/(Xid-xid+1))+yid;
		_x=i%(Xid-xid+1)+xid;
		access=_x+_y*sidenum+_z*sidenum*sidenum;
		for(j=0;j<count[access];j++) M[BEG[i]+j]=newP[begin[access]+j];
	}
	__syncthreads();
	step=ceil(float(totalnum/threadnum));
	for(i=0;i<totalnum;i++) if(
		(M[i].x-_POINT.x)*(M[i].x-_POINT.x)+
		(M[i].y-_POINT.y)*(M[i].y-_POINT.y)+
		(M[i].z-_POINT.z)*(M[i].z-_POINT.z)<=R*R )
		sum+=M[i].w;
	ans[id]=sum;
	__syncthreads();
}
__global__ void RESORT(int N,int* retrive,float* ANS,float* ans, int threadnum, int blocknum) {
	int i;
	int tid,bid,id,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=bid*threadnum+tid;
	step=ceil(float(N)/blocknum/threadnum);
	for(i=id*step;i<N && i<(id+1)*step;i++) 
		ANS[retrive[i]]=ans[i];
	__syncthreads();
}
void N_BODY(int _N, const float4* P, float r, float R)
{
	int i,N=_N;
	float4 *oldP,*newP;
	float *ans,*ANS;
	int *count,*belong,*retrive,*begin;

	int threadnum, blocknum, sidenum, cubenum;
	float OUT[MAX];

	threadnum=64;
	blocknum=N/threadnum;
	sidenum=ceil(1.0/r);
	cubenum=sidenum*sidenum*sidenum;
	hipMalloc((void**)&ans,N*sizeof(float));
	hipMalloc((void**)&ANS,N*sizeof(float));
	hipMalloc((void**)&oldP,N*sizeof(float4));
	hipMalloc((void**)&newP,N*sizeof(float4));
	hipMalloc((void**)&count,cubenum*sizeof(int));	
	hipMalloc((void**)&begin,cubenum*sizeof(int));
	hipMalloc((void**)&belong,N*sizeof(int));
	hipMalloc((void**)&retrive,N*sizeof(int));
	hipMemcpy(oldP,P,N*sizeof(float4),hipMemcpyHostToDevice);
	SORT(N,count,begin,belong,retrive,oldP,newP,threadnum, blocknum, sidenum, cubenum, r);
	//the last parameter is used for the settlement of insufficient shared memory
	//if the shared memory is not enough, we simply cut the particals to several grids
	//each grids could use the total number of shared memory, and the blocks contained 
	//in each grid should be maxized thus the number of grids is minizied
	//successively grids were operately seriesly
	CALC<<<blocknum,threadnum>>>(newP,ans,count,begin,0,threadnum, blocknum, sidenum, r, R);
	RESORT<<<blocknum,threadnum>>>(N,retrive,ANS,ans,threadnum, blocknum);
	hipMemcpy(OUT,ANS,N*sizeof(float),hipMemcpyDeviceToHost);
	for(i=0;i<N;i++) printf("%f\n",OUT[i]);
}
void input(int *N, float *R, float4 *P) {
	int i;
	FILE* file=fopen("input.txt","r");
	fscanf(file,"%d%f",N,R);
	for(i=0;i<*N;i++) fscanf(file,"%f%f%f%f",&(P[i].x),&(P[i].y),&(P[i].z),&(P[i].w));
}
int main(int argc, char** argv)
{
		float r=0.1;
		float R;
		int N;
		float4 P[MAX];


	freopen("output.txt","w",stdout);
	if(!InitCUDA()) return 0;
	printf("CUDA initialized.\n");
	input(&N, &R, P);
	N_BODY(N,P,r,R);
	return 0;
}
