#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

/*
 * WORLD_SIZE	The length of the whole world's edge
 * R			The length of the seperated cubes' edge
 * CUBE_NUM		Total number of seperated cubes(should equal to (WORLD_SIZE^3 / R^3)
 * CUBE_PER_EDGE	Cubes on one edge (WORLD_SIZE / R)
 */
#define WORLD_SIZE 1
#define R 0.1
#define CUBE_NUM 1000
#define CUBE_PER_EDGE 10

struct axis {
	float x;
	float y;
	float z;
	float v;
};

struct cube_info {
	int start;
	int length;
};

int cal(FILE *fp);

int main(int argc, char *argv[])
{
	FILE *fp;

	if (argc >= 2) {
		fp = fopen(argv[1], "r");

	} else {
		printf("Usage: prog datafile\n");
		exit(1);
	}

	cal(fp);
	fclose(fp);

	return 0;
}


__device__ float distance(struct axis *p1, struct axis *p2)
{
	return (p1->x - p2->x) * (p1->x - p2->x) +
		(p1->y - p2->y) * (p1->y - p2->y) +
		(p1->z - p2->z) * (p1->z - p2->z);
}

__global__ void threadCode(int count, float radius2, struct axis *points, float *result)
{
	int thId;
	int i;
	/*float tmp_rst;*/
	int base_idx;
	int tmp_idx;

	__shared__ struct axis block_elem[BLOCK_SIZE];
	__shared__ struct axis sh_data[SHARE_CACHE];
	__shared__ float sh_rst[BLOCK_SIZE];

	thId = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	sh_rst[threadIdx.x] = 0;

	for (base_idx = 0; base_idx <= count; base_idx += SHARE_CACHE) {

		int up_lim = count - base_idx;
		up_lim = up_lim>SHARE_CACHE ? SHARE_CACHE : up_lim;

		__syncthreads();

		/*tmp_idx = threadIdx.x*2;*/
		/*sh_data[tmp_idx] = points[base_idx+tmp_idx];*/
		/*++tmp_idx;*/
		/*sh_data[tmp_idx] = points[base_idx+tmp_idx];*/
		if (threadIdx.x == 0) {
			for (tmp_idx = 0; tmp_idx < SHARE_CACHE; tmp_idx++) {
				sh_data[tmp_idx] = points[base_idx+tmp_idx];
			}
		}
		__syncthreads();

		if (thId >= count)
			return;
		block_elem[threadIdx.x] = points[thId];


		/*tmp_rst = 0;*/
		for (i = 0; i < up_lim; i++) {
			if ((base_idx+i) != thId &&
					distance(&block_elem[threadIdx.x], &sh_data[i]) <= radius2) {
				sh_rst[threadIdx.x] += sh_data[i].v;
			}
		}
	}
	result[thId] = sh_rst[threadIdx.x];
}

int paralize(int count, float radius, struct axis *points, float *sum)
{
	float *cudaRst;
	struct axis *cudaPtr;

	hipMalloc((void **)&cudaPtr, sizeof(struct axis)*count);
	hipMalloc((void **)&cudaRst, sizeof(float)*count);

	hipMemcpy(cudaPtr, points, sizeof(struct axis)*count, hipMemcpyHostToDevice);
	hipMemset(cudaRst, 0x0, sizeof(float)*count);

	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid((count+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

	threadCode<<<dimGrid, dimBlock>>>(count, radius*radius, cudaPtr, cudaRst);
	
	hipMemcpy(sum, cudaRst, sizeof(float)*count, hipMemcpyDeviceToHost);
	hipFree(cudaPtr);
	hipFree(cudaRst);
	return 0;
}

int cal(FILE *fp)
{
	int point_count;
	float radius;
	struct axis *points;
	int i;
	struct timeval tv_start, tv_end;
	double time_cost;
	float *sum;

	fscanf(fp, "%d %f", &point_count, &radius);

	points = (struct axis *) malloc(sizeof(struct axis)*point_count);
	sum = (float *) malloc(sizeof(float)*point_count);
	memset((void *) sum, 0x0, sizeof(float)*point_count);

	for (i = 0; i < point_count; i++)
		fscanf(fp, "%f %f %f %f", &points[i].x,	&points[i].y, 
				&points[i].z, &points[i].v);

	/* execute calculation and get time stamp */
	gettimeofday(&tv_start, NULL);
	paralize(point_count, radius, points, sum);
	gettimeofday(&tv_end, NULL);

	time_cost = 1000000 * (tv_end.tv_sec - tv_start.tv_sec) +
		(tv_end.tv_usec - tv_start.tv_usec);
	time_cost /= 1000000;

	/* output the result and time cost */
	for (i = 0; i < point_count; i++)
		printf("Point %5d: %f\n", i+1, sum[i]);
	printf("Time: %lf\n", time_cost);

	free(points);
	free(sum);
	return 0;
}
