#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;
	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
/************************************************************************/
float r=0.1;
const int MAX=1000000;
int threadnum,blocknum,sidenum,N,cubenum;
float R,OUT[MAX];
float4 P[MAX];
__global__ void MEMSET(int size,int* arr,int value) {
	int i,j,k,tid,bid,id,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=bid*threadnum+tid;
	step=ceil(float(size)/threadnum/blocknum);
	for(i=step*id;i<size && i<step*(id+1);i++) arr[i]=value;
	__syncthreads();
}
__device__ void device_MEMSET(int size,int* arr,int value) {
	int i,j,k,tid,step;
	tid=threadIdx.x;
	step=ceil(float(size)/threadnum);
	for(i=step*tid;i<size && i<step*(tid+1);i++) arr[i]=value;
	__syncthreads();
}
__global__ void SORT_1(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP) {
	int i,j,k,tid,bid,id,IDx,IDy,IDz,ID,beg,end,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	step=ceil(float(N)/blocknum/threadnum);
	beg=step*id;
	end=beg+step;
	for(i=beg;i<N && i<end;i++) {
		IDx=oldP[i].x/r;	IDy=oldP[i].y/r;	IDz=oldP[i].z/r;
		ID=IDx+IDy*sidenum+IDz*sidenum*sidenum;
		belong[i]=ID;	begin[ID]++;	count[ID]++;
	}
}
__global__ void SORT_2(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP) {
	int i,j,k,tid,bid,id;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	if(id==0) for(i=1;i<cubenum;i++) begin[i]+=begin[i-1];
}
__global__ void SORT_3(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP) {
	int i,j,k,tid,bid,id,beg,end,step,ID;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=tid+bid*threadnum;
	step=ceil(float(N)/blocknum/threadnum);
	beg=step*id;
	end=beg+step;
	for(i=beg;i<end;i++) { 
		ID=belong[i];
		j=--begin[ID];
		retrive[j]=i;
		newP[j]=oldP[i];
	}
}
void SORT(int N,int* count,int* begin,int* belong,int* retrive,float4* oldP,float4* newP) {
	MEMSET<<<blocknum,threadnum>>>(cubenum,begin,0);
	MEMSET<<<blocknum,threadnum>>>(cubenum,count,0);
	SORT_1<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP);
	SORT_2<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP);
	SORT_3<<<blocknum,threadnum>>>(N,count,begin,belong,retrive,oldP,newP);
}
//make sure that threadnum*blocknum equals to the number of total particals
__global__ void CALC(float4* newP,float* ans,int* count,int* begin) {
	int i,j,k,tid,bid,id,beg,end,step;
	__shared__ float max_x,max_y,max_z,min_x,min_y,min_z;
	float sum=0;
	__shared__ int xid,Xid,yid,Yid,zid,Zid,readnum,totalnum;
	int access,_x,_y,_z;
	float4 _POINT;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=bid*threadnum+tid;
	max_x=max_y=max_z=-1, min_x=min_y=min_z=1;
	__syncthreads();
	//every threads calculate one partical
	//figure out the perimeter of the zone in the block
	_POINT=newP[id];
	if(max_x<_POINT.x) max_x=_POINT.x;
	if(max_y<_POINT.y) max_y=_POINT.y;
	if(max_z<_POINT.z) max_z=_POINT.z;
	if(min_x>_POINT.x) min_x=_POINT.x;
	if(min_y>_POINT.y) min_y=_POINT.y;
	if(min_z>_POINT.z) min_z=_POINT.z;
	if(min_x-R>0) min_x-=R;	else min_x=0;
	if(min_y-R>0) min_y-=R;	else min_y=0;
	if(min_z-R>0) min_z-=R; else min_z=0;
	if(max_x+R<1) max_x+=R;	else max_x=0.99998;
	if(max_y+R<1) max_y+=R;	else max_y=0.99998;
	if(max_z+R<1) max_z+=R; else max_z=0.99998;
	xid=min_x/r,yid=min_y/r,zid=min_z/r;
	Xid=max_x/r,Yid=max_y/r,Zid=max_z/r;
	__syncthreads();
	readnum=(Xid-xid+1)*(Yid-yid+1)*(Zid-zid+1);
	__shared__ float4 M[MAX];
	__shared__ int BEG[MAX];
	device_MEMSET(readnum,BEG,0);
	if(tid==0) 	{
		for(i=0;i<readnum;i++) {
			_z=int(i/(Xid-xid+1)/(Yid-yid+1))+xid;
			_y=int((i-_z*(Xid-xid+1)*(Yid-yid+1))/(Xid-xid+1))+yid;
			_x=i%(Xid-xid+1)+xid;
			access=_x+_y*sidenum+_z*sidenum*sidenum;
			BEG[i+1]=BEG[i]+count[access];
		}
		totalnum=BEG[readnum];
	}
	__syncthreads();
	step=ceil(float(readnum)/threadnum);
	for(i=tid*step;i<readnum && i<(tid+1)*step;i++) {
		_z=int(i/(Xid-xid+1)/(Yid-yid+1))+xid;
		_y=int((i-_z*(Xid-xid+1)*(Yid-yid+1))/(Xid-xid+1))+yid;
		_x=i%(Xid-xid+1)+xid;
		access=_x+_y*sidenum+_z*sidenum*sidenum;
		for(j=0;j<count[access];j++) M[BEG[i]+j]=newP[begin[access]+j];
	}
	__syncthreads();
	step=ceil(float(totalnum/threadnum));
	for(i=0;i<totalnum;i++) if(
		(M[i].x-_POINT.x)*(M[i].x-_POINT.x)
		+(M[i].y-_POINT.y)*(M[i].y-_POINT.y)
		+(M[i].z-_POINT.z)*(M[i].z-_POINT.z)<=R*R )
		sum+=M[i].w;
	ans[id]=sum;
	__syncthreads();
}
__global__ void RESORT(int N,int* retrive,float* ANS,float* ans) {
	int i,j,k;
	int tid,bid,id,totalnum,step;
	tid=threadIdx.x;
	bid=blockIdx.x;
	id=bid*threadnum+tid;
	step=ceil(float(N)/blocknum/threadnum);
	for(i=id*step;i<N && i<(id+1)*step;i++) 
		ANS[retrive[i]]=ans[i];
	__syncthreads();
}
void N_BODY(int _N, const float4* P)
{
	int i,j,k,N=_N;
	float4 *oldP,*newP;
	float *ans,*ANS;
	int *count,*belong,*retrive,*begin;
	threadnum=64;
	blocknum=N/threadnum;
	sidenum=ceil(1.0/r);
	cubenum=sidenum*sidenum*sidenum;
	hipMalloc((void**)&ans,N*sizeof(float));
	hipMalloc((void**)&ANS,N*sizeof(float));
	hipMalloc((void**)&oldP,N*sizeof(float4));
	hipMalloc((void**)&newP,N*sizeof(float4));
	hipMalloc((void**)&count,cubenum*sizeof(int));	
	hipMalloc((void**)&begin,cubenum*sizeof(int));
	hipMalloc((void**)&belong,N*sizeof(int));
	hipMalloc((void**)&retrive,N*sizeof(int));
	hipMemcpy(oldP,P,N*sizeof(float4),hipMemcpyHostToDevice);
	SORT(N,count,begin,belong,retrive,oldP,newP);
	CALC<<<blocknum,threadnum>>>(newP,ans,count,begin);
	RESORT<<<blocknum,threadnum>>>(N,retrive,ANS,ans);
	hipMemcpy(OUT,ANS,N*sizeof(float),hipMemcpyDeviceToHost);
	for(i=0;i<N;i++) printf("%f\n",OUT[i]);
}
void input() {
	int i,j,k;
	FILE* file=fopen("input.txt","r");
	fscanf(file,"%d %f",&N,&R);
	for(i=0;i<N;i++) fscanf(file,"%f%f%f%f",&(P[i].x),&(P[i].y),&(P[i].z),&(P[i].w));
}
int main(int argc, char** argv)
{
	if(!InitCUDA()) return 0;
	printf("CUDA initialized.\n");
	input();
	N_BODY(N,P);
	return 0;
}
