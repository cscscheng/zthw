// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
void randomInit(float* data, int size);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len, const unsigned int dimension);
bool hComparef( const float* reference, const float* data, const unsigned int len);

// includes, kernels
#include <knn_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	printf("particle num is:%d \n",particle_num);
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//runTest
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

	srand((unsigned int)time(0));

	unsigned int i_data_size = particle_num * (particle_dimension + 1);		//�������������������С �� ��������������ά����1��
	unsigned int i_mem_size = sizeof( float) * i_data_size;					//���ڴ洢������������ռ��ֽ���				
	unsigned int o_mem_size = sizeof( float) * particle_num;				//���ڴ洢�������ռ��ֽ���

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    // ���� host memory
    float* h_idata = (float*) malloc( i_mem_size);		//�������ݿռ���h_idata��
    
    randomInit(h_idata, i_data_size);					//�����������ݷ���h_idata


    // ����device memory
    float* d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, i_mem_size));

    // ���ڴ��е�h_idata�������Դ��е�d_idata��ȥ
    cutilSafeCall( hipMemcpy( d_idata, h_idata, i_mem_size,
                                hipMemcpyHostToDevice) );

    // ���� device memory ���ڴ洢���
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, o_mem_size));

    // �����ں�Grid��Block����һά��
    dim3  grid( block_num, 1, 1);
    dim3  threads( thread_num, 1, 1);

    // ִ��kernel
    testKernel<<< grid, threads >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // �����ڴ�ռ�洢device�ϼ���Ľ��
    float* h_odata = (float*) malloc( o_mem_size);
    
	// ��device�ϵĽ��������host�ϵ�h_odata������ȥ
    cutilSafeCall( hipMemcpy( h_odata, d_odata, o_mem_size,
                                hipMemcpyDeviceToHost) );


    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time in GPU: %f (ms)\n", cutGetTimerValue( timer));

	/////////////////////////////////////////////////////////////////////////////
	//////CPU�ϵļ���
	////////////////////////////////////////////////////////////////////////////
    cutilCheckError( cutStartTimer( timer));
	
	// �����ڴ�ռ����ڴ洢CPU�ϵļ�����
    float* reference = (float*) malloc( o_mem_size);

    computeGold( reference, h_idata, particle_num, particle_dimension);

	cutilCheckError( cutStopTimer( timer));
    printf( "Processing time in CPU: %f (ms)\n", cutGetTimerValue( timer));
    
	cutilCheckError( cutDeleteTimer( timer));

    bool res = hComparef( reference, h_odata, particle_num);
    printf( "\n GPU����CPU�ϲ��Խ���Ƿ�ͨ���� %s\n", (1 == res) ? "PASSED" : "FAILED");

    // �����ڴ�ռ�
    free( h_idata);
    free( h_odata);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}

//����������ݺ���
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
      data[i] = rand() / (float)RAND_MAX;
}

//�Ƚϲ��Խ��������������
bool hComparef( const float* reference, const float* data, const unsigned int len)
{        
        bool result = true;
		int error_num;
		error_num = 0;

        for( unsigned int i = 0; i < len; ++i) {

            float diff = reference[i] - data[i];
            bool comp = (diff <= 0.0f) && (diff >= -0.0f);
            result &= comp;

			if( ! comp) 
				error_num++;
        }
		printf("ERROR_NUM = %d ,\n", error_num);
        return (result) ? true : false;
}